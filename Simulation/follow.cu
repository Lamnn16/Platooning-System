#include "hip/hip_runtime.h"
#include "follow.h"
#include <hip/hip_runtime.h>

FollowingVehicle::FollowingVehicle(int id, double initialSpeed, double initialPosition, double targetDistance, double Kp, double Ki, double Kd)
    : id(id), position(initialPosition), speed(initialSpeed), serverSocket(0), targetDistance(targetDistance),
      Kp(Kp), Ki(Ki), Kd(Kd), integralError(0.0), previousError(0.0) {}

int FollowingVehicle::getId() const
{
    return id;
}

double FollowingVehicle::getPosition() const
{
    return position;
}

double FollowingVehicle::getSpeed() const
{
    return speed;
}

void FollowingVehicle::setPosition(double newPosition)
{
    position = newPosition;
}

void FollowingVehicle::setSpeed(double newSpeed)
{
    speed = newSpeed;
}

void FollowingVehicle::connectToLeader(const std::string &ipAddress)
{
    // Create client socket and connect to the leading vehicle
    serverSocket = socket(AF_INET, SOCK_STREAM, 0);
    if (serverSocket < 0)
    {
        std::cerr << "Error creating client socket" << std::endl;
        return;
    }

    struct sockaddr_in serverAddress;
    serverAddress.sin_family = AF_INET;
    serverAddress.sin_port = htons(PORT);

    if (inet_pton(AF_INET, ipAddress.c_str(), &(serverAddress.sin_addr)) <= 0)
    {
        std::cerr << "Invalid address/ Address not supported" << std::endl;
        return;
    }

    if (connect(serverSocket, (struct sockaddr *)&serverAddress, sizeof(serverAddress)) < 0)
    {
        std::cerr << "Connection failed" << std::endl;
        return;
    }
    std::cout << "Connected to the leading vehicle!" << std::endl;
}

void FollowingVehicle::sendFollowerMessage()
{
    // Send follower message
    Message followerMessage;
    followerMessage.senderId = id;
    followerMessage.position = position;
    followerMessage.speed = speed;
    followerMessage.isConnected = true;

    if (send(serverSocket, &followerMessage, sizeof(followerMessage), 0) < 0)
    {
        std::cerr << "Error sending follower message" << std::endl;
    }
}

void FollowingVehicle::sendFollowerMessagesContinuously(int interval)
{
    while (true)
    {
        sendFollowerMessage();
        std::this_thread::sleep_for(std::chrono::milliseconds(interval));
    }
}

// CUDA kernel function for calculating speed and distance
__global__ void calculateSpeedAndDistance(double* deviceSpeed, double* devicePosition, double leaderPosition,double targetSpeed, double targetDistance, double Kp, double Ki, double Kd)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate speed
    double error = targetSpeed - deviceSpeed[index];
    double integralError = 0.0; // Assume integralError and previousError are stored in device memory
    double previousError = 0.0;
    double derivativeError = error - previousError;
    double controlSignal = Kp * error + Ki * integralError + Kd * derivativeError;
    deviceSpeed[index] += controlSignal;

    // Calculate distance
    double distanceError = targetDistance - (leaderPosition - devicePosition[index]);
    double positionControlSignal = Kp * distanceError;
    devicePosition[index] -= positionControlSignal;
}

void FollowingVehicle::receiveStateFromLeader()
{
    Message message;

    if (recv(serverSocket, &message, sizeof(message), 0) < 0)
    {
        std::cerr << "Error receiving state from leader" << std::endl;
    }
    else
    {
        // Use the received speed as the targetSpeed
        double targetSpeed = message.speed;
        double leaderPosition = message.position;

        // Allocate device memory for speed and position
        double* deviceSpeed;
        double* devicePosition;
        hipMalloc((void**)&deviceSpeed, sizeof(double));
        hipMalloc((void**)&devicePosition, sizeof(double));

        // Copy speed and position data to device memory
        hipMemcpy(deviceSpeed, &speed, sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(devicePosition, &position, sizeof(double), hipMemcpyHostToDevice);

        // Call the CUDA kernel for speed and distance calculation
        calculateSpeedAndDistance<<<1, 1>>>(deviceSpeed, devicePosition, leaderPosition, targetSpeed, targetDistance, Kp, Ki, Kd);

        // Copy the results back to the host
        hipMemcpy(&speed, deviceSpeed, sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(&position, devicePosition, sizeof(double), hipMemcpyDeviceToHost);

        // Free device memory
        hipFree(deviceSpeed);
        hipFree(devicePosition);
    }
}

void FollowingVehicle::receiveMessageFromLeader()
{
    while (true)
    {
        receiveStateFromLeader();
        std::this_thread::sleep_for(std::chrono::milliseconds(100)); // Add a delay between receiving messages
    }
}

void FollowingVehicle::printState()
{
    std::cout << "Following Vehicle ID: " << id;
    std::cout << "  Position: " << position;
    std::cout << "  Speed: " << speed << std::endl;
}

void FollowingVehicle::stopServer()
{
    close(serverSocket);
}

// Function to set the terminal in non-canonical mode and disable input buffering
void setNonCanonicalMode()
{
    struct termios t;
    tcgetattr(STDIN_FILENO, &t);
    t.c_lflag &= ~(ICANON | ECHO);
    tcsetattr(STDIN_FILENO, TCSANOW, &t);
}

// Function to restore the terminal settings
void restoreTerminalSettings()
{
    struct termios t;
    tcgetattr(STDIN_FILENO, &t);
    t.c_lflag |= ICANON | ECHO;
    tcsetattr(STDIN_FILENO, TCSANOW, &t);
}

// Function to check if there is keyboard input available
bool isKeyPressed()
{
    fd_set readSet;
    FD_ZERO(&readSet);
    FD_SET(STDIN, &readSet);

    struct timeval timeout;
    timeout.tv_sec = 0;
    timeout.tv_usec = 0;

    int selectResult = select(STDIN + 1, &readSet, NULL, NULL, &timeout);
    if (selectResult == -1)
    {
        std::cerr << "Error in select" << std::endl;
        return false;
    }

    return FD_ISSET(STDIN, &readSet);
}

int main(int argc, char *argv[])
{
    // Default values
    double initialSpeed = 0.0;
    double targetDistance = 10.0;
    double Kp = 0.1;
    double Ki = 0.01;
    double Kd = 0.01;
    int id = 2;
    double initialPosition = 10.0;

    // Parse command-line arguments
    for (int i = 1; i < argc; ++i)
    {
        std::string arg = argv[i];
        if (arg == "--id" && i + 1 < argc)
        {
            id = std::stoi(argv[i + 1]);
        }
        if (arg == "--initspeed" && i + 1 < argc)
        {
            initialSpeed = std::stod(argv[i + 1]);
        }
        else if (arg == "--initposition" && i + 1 < argc)
        {
            initialPosition = std::stod(argv[i + 1]);
        }
        else if (arg == "--distance" && i + 1 < argc)
        {
            targetDistance = std::stod(argv[i + 1]);
        }
    }

    FollowingVehicle follower(id, initialSpeed, initialPosition, targetDistance, Kp, Ki, Kd);

    std::string ipAddress = "127.0.0.1"; // IP address of the leading vehicle

    // Connect to the leader
    follower.connectToLeader(ipAddress);

    // Start a separate thread to receive messages from the leader continuously
    std::thread receiveThread(&FollowingVehicle::receiveMessageFromLeader, &follower);
    receiveThread.detach(); // Detach the thread to let it run independently

    // Start a separate thread to continuously send follower messages
    std::thread followerMessageThread(&FollowingVehicle::sendFollowerMessagesContinuously, &follower, 1000); // Send follower message every 1 second
    followerMessageThread.detach();                                                                          // Detach the thread to let it run independently

    setNonCanonicalMode(); // Set the terminal in non-canonical mode
    while (true)
    {
        follower.printState();

        // Check for input
        if (isKeyPressed())
        {
            char c;
            std::cin.get(c);
            if (c == 'q')
            {
                std::cout << "The Following Vehicle has left the platoon " << std::endl;
                break;
            }
        }

        sleep(1);
    }

    restoreTerminalSettings(); // Restore the terminal settings

    follower.stopServer();

    return 0;
}